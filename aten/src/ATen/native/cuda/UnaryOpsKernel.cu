#include "hip/hip_runtime.h"
#include <limits>
#include <ATen/native/UnaryOps.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/Context.h>
#include <ATen/Dispatch.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/TensorIterator.h>

namespace at { namespace native {

void bitwise_not_kernel_cuda(TensorIterator& iter) {
  if (iter.dtype() == ScalarType::Bool) {
    gpu_kernel(iter, []GPU_LAMBDA(bool a) {
      return !a;
    });
  } else {
    AT_DISPATCH_INTEGRAL_TYPES(iter.dtype(), "bitwise_not_cuda", [&]() {
      gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
        return ~a;
      });
    });
  }
}

void logical_not_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_ALL_TYPES_AND2(kBool, kHalf, iter.dtype(1), "logical_not_cuda", [&]() {
    using self_t = scalar_t;
    AT_DISPATCH_ALL_TYPES_AND2(kBool, kHalf, iter.dtype(0), "logical_not_cuda", [&]() {
      gpu_kernel(iter, []GPU_LAMBDA(self_t a) -> scalar_t { return static_cast<scalar_t>(!a); });
    });
  });
}

void ceil_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "ceil_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return std::ceil(a);
    });
  });
}

void neg_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_ALL_TYPES_AND(ScalarType::Half, iter.dtype(), "neg_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return -a;
    });
  });
}

void sign_kernel_cuda(TensorIterator& iter){
    if (iter.dtype() == ScalarType::Bool) {
      gpu_kernel(iter, []GPU_LAMBDA(bool a){
        return a;
      });
    } else {
      AT_DISPATCH_ALL_TYPES_AND(ScalarType::Half, iter.dtype(), "sign_cuda", [&]() {
          gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
              scalar_t zero = scalar_t(0);
              return (zero < a) - (a < zero);
          });
      });
    }
}

void erfinv_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "erfinv_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return erfinvf(a);
    });
  });
}

REGISTER_DISPATCH(bitwise_not_stub, &bitwise_not_kernel_cuda);
REGISTER_DISPATCH(logical_not_stub, &logical_not_kernel_cuda);
REGISTER_DISPATCH(ceil_stub, &ceil_kernel_cuda);
REGISTER_DISPATCH(neg_stub, &neg_kernel_cuda);
REGISTER_DISPATCH(sign_stub, &sign_kernel_cuda);
REGISTER_DISPATCH(erfinv_stub, &erfinv_kernel_cuda);
}}
