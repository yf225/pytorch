#include "hip/hip_runtime.h"
#include <limits>
#include <ATen/native/UnaryOps.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/Context.h>
#include <ATen/Dispatch.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/cuda/Math.cuh>

namespace at { namespace native {

void bitwise_not_kernel_cuda(TensorIterator& iter) {
  if (iter.dtype() == ScalarType::Bool) {
    gpu_kernel(iter, []GPU_LAMBDA(bool a) {
      return !a;
    });
  } else {
    AT_DISPATCH_INTEGRAL_TYPES(iter.dtype(), "bitwise_not_cuda", [&]() {
      gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
        return ~a;
      });
    });
  }
}

void logical_not_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_ALL_TYPES_AND2(kBool, kHalf, iter.dtype(1), "logical_not_cuda", [&]() {
    using self_t = scalar_t;
    AT_DISPATCH_ALL_TYPES_AND2(kBool, kHalf, iter.dtype(0), "logical_not_cuda", [&]() {
      gpu_kernel(iter, []GPU_LAMBDA(self_t a) -> scalar_t { return static_cast<scalar_t>(!a); });
    });
  });
}

void ceil_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "ceil_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return std::ceil(a);
    });
  });
}

void neg_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_ALL_TYPES_AND(ScalarType::Half, iter.dtype(), "neg_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return -a;
    });
  });
}

void sign_kernel_cuda(TensorIterator& iter){
    if (iter.dtype() == ScalarType::Bool) {
      gpu_kernel(iter, []GPU_LAMBDA(bool a){
        return a;
      });
    } else {
      AT_DISPATCH_ALL_TYPES_AND(ScalarType::Half, iter.dtype(), "sign_cuda", [&]() {
          gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
              scalar_t zero = scalar_t(0);
              return (zero < a) - (a < zero);
          });
      });
    }
}

void erfinv_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "erfinv_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return erfinvf(a);
    });
  });
}

void digamma_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "digamma_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return calc_digamma(a);
    });
  });
}

void trigamma_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "trigamma_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return calc_trigamma(a);
    });
  });
}

void polygamma_kernel_cuda(TensorIterator& iter, int64_t n) {
  switch (n) {
    case 0: digamma_kernel_cuda(iter); break;
    case 1: trigamma_kernel_cuda(iter); break;
    default: TORCH_CHECK(false, "polygamma(n,x) is not implemented for n>=2, but was ", n);
  }
}

REGISTER_DISPATCH(bitwise_not_stub, &bitwise_not_kernel_cuda);
REGISTER_DISPATCH(logical_not_stub, &logical_not_kernel_cuda);
REGISTER_DISPATCH(ceil_stub, &ceil_kernel_cuda);
REGISTER_DISPATCH(neg_stub, &neg_kernel_cuda);
REGISTER_DISPATCH(sign_stub, &sign_kernel_cuda);
REGISTER_DISPATCH(erfinv_stub, &erfinv_kernel_cuda);
REGISTER_DISPATCH(digamma_stub, &digamma_kernel_cuda);
REGISTER_DISPATCH(polygamma_stub, &polygamma_kernel_cuda);
}}
